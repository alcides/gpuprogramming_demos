
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void add(int a, int b, int *c) {
    *c = a + b;
}

int main( void ) {
    int c, *dev_c;
	hipMalloc(&dev_c, sizeof(int));
    add<<<1, 1>>>(2, 7, dev_c);
	hipMemcpy(&c, dev_c, sizeof(int), hipMemcpyDeviceToHost);
	hipFree(dev_c);
	printf("2 + 7 = %d\n", c);
}
