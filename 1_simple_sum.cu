
#include <hip/hip_runtime.h>
#include <stdio.h>

#define N 10

__global__ void sum(int *a,
         int *b, int *c)
{
  int i;
  for(i = 0; i < N; i++) {
     c[i] = a[i] + b[i];
  }
}


int main( void ) {

	int host_a[N];
	int host_b[N];
	int host_c[N];
	
	for (int i=0; i<N; i++) {
		host_a[i] = i;
		host_b[i] = i;
	}

	int *dev_a, *dev_b, *dev_c;
	hipMalloc(&dev_a, sizeof(int) * N);
	hipMalloc(&dev_b, sizeof(int) * N);
	hipMalloc(&dev_c, sizeof(int) * N);
	hipMemcpy(dev_a, host_a, sizeof(int) * N, hipMemcpyHostToDevice);
	hipMemcpy(dev_b, host_b, sizeof(int) * N, hipMemcpyHostToDevice);

	sum<<<1, 1>>>(dev_a, dev_b, dev_c);

	hipMemcpy(host_c, dev_c, sizeof(int) * N, hipMemcpyDeviceToHost);
	
	for (int i=0; i<N; i++) {
		printf("%d ", host_c[i]);
	}
	printf("\n");

}
