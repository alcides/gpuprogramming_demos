
#include <hip/hip_runtime.h>
#include <stdio.h>

#define N 3000

__global__ void sum(int *a, int *b, int *c) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  while (i < N) {
    c[i] = a[i] + b[i];
    i += gridDim.x * blockDim.x;
  }
}


int main( void ) {

	int host_a[N];
	int host_b[N];
	int host_c[N];
	
	for (int i=0; i<N; i++) {
		host_a[i] = i;
		host_b[i] = i;
	}

	int *dev_a, *dev_b, *dev_c;
	hipMalloc(&dev_a, sizeof(int) * N);
	hipMalloc(&dev_b, sizeof(int) * N);
	hipMalloc(&dev_c, sizeof(int) * N);
	hipMemcpy(dev_a, host_a, sizeof(int) * N,
	         hipMemcpyHostToDevice);
	hipMemcpy(dev_b, host_b, sizeof(int) * N,
	         hipMemcpyHostToDevice);
		 
	sum<<<20, 30>>>(dev_a, dev_b, dev_c);

	hipMemcpy(host_c, dev_c, sizeof(int) * N,
	            hipMemcpyDeviceToHost);
	for (int i=0; i<N; i++) {
		printf("%d ", host_c[i]);
	}
	printf("\n");


}


